#include "hip/hip_runtime.h"
#include "cuda_funcs.h"
#include "cuda_err_check.hpp"

constexpr int FLOAT_2048 = 0xc5000000;      // when convert to float directly, this is equal to 2048.0
__constant__ float raw_segs[2048];

__host__ void copyRawSegs(const float* const host_segs, size_t byte_num) {
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(raw_segs), host_segs, byte_num, 0, hipMemcpyHostToDevice));
}

__device__ __forceinline__ int floatToOrderedInt( float floatVal ) {
 int intVal = __float_as_int( floatVal );
    return (intVal >= 0 ) ? intVal ^ 0x80000000 : intVal ^ 0xFFFFFFFF;
}

__device__ __forceinline__ float orderedIntToFloat( int intVal ) {
    return __int_as_float( (intVal >= 0) ? intVal ^ 0xFFFFFFFF : intVal ^ 0x80000000);
}

__device__ void initialize(const float* const segments, const Eigen::Vector2d* const obs, int id, bool* flags) {
    const int base = 4 * id;
    const Eigen::Vector2d pt1(*(segments + base), *(segments + base + 1)), pt2(*(segments + base + 2), *(segments + base + 3));
    const Eigen::Vector2d norm(pt1.y() - pt2.y(), pt2.x() - pt1.x());
    const Eigen::Vector2d ctr_vec = (pt1 + pt2) / 2.0 - Eigen::Vector2d(obs->x(), obs->y());
    if (ctr_vec.x() * norm.x() + ctr_vec.y() * norm.y() > 0.0)
        flags[id] = false;
    else flags[id] = true;
}

template<bool singl>
__device__ bool isIdInRange(const int range_sid, const int range_eid, const int id) {
    if (singl == true)
        return (id >= range_sid) || (id <= range_eid);
    else
        return (id >= range_sid) && (id <= range_eid);
}

template<bool singl>
__device__ int getRangeOffset(const int range_sid, const int range_eid, const int id, const int range_num) {
    if (singl == true)
        if (id <= range_eid)
            return max(range_num - range_eid + id - 1, 0);
    return min(id - range_sid, range_num - 1);
}

__device__ float getRange(const Eigen::Vector2d& p1, const Eigen::Vector2d& vec_line, const Eigen::Vector2d& obs_pt, const double angle) {
    const Eigen::Vector2d vec(cos(angle), sin(angle));
    Eigen::Matrix2d A;
    A << vec_line(0), vec_line(1), -vec(0), -vec(1);
    const double b1 = -vec(1) * obs_pt(0) + vec(0) * obs_pt(1);
    const double b2 = -vec_line(1) * p1(0) + vec_line(0) * p1(1);
    const Eigen::Vector2d b(b1, b2);
    const double det = A(0, 0) * A(1, 1) - A(0, 1) * A(1, 0);
    if (std::abs(det) < 1e-5) {
        return (p1 - obs_pt).norm();
    }
    A /= det;
    return (A * b - obs_pt).norm();
}

__device__ void singleSegZbuffer(
    const Eigen::Vector2d& p1, const Eigen::Vector2d& p2, const Obsp* const ptcls,
    const int s_id, const int e_id, const int range_num,
    const double ang_incre, int* range
) {
    const Eigen::Vector2d obs(ptcls->x, ptcls->y);
    const Eigen::Vector2d ray1 = p1 - obs, ray2 = p2 - obs, vec_line = p2 - p1;
    const double sang = atan2(ray1(1), ray1(0)), eang = atan2(ray2(1), ray2(0));
    const int id_s = static_cast<int>(ceil((sang + M_PI) / ang_incre)),
        id_e = static_cast<int>(floor((eang + M_PI) / ang_incre));
    if (id_s == id_e + 1) {
        return;
    }
    const int max_ray_num = round(2 * M_PI / ang_incre);
    bool not_int_range = false;
    const bool range_singl = (s_id > e_id), edge_singl = (id_s > id_e);
    if (range_singl) {          // 深度图范围角度奇异
        if (edge_singl == false) {      // 被投影边角度不奇异
            if (id_e < s_id && id_s > e_id) {
                not_int_range = true;
            }
        }                       // 奇异必定有重合部分
    } else {
        if (edge_singl) {      // 被投影边角度奇异
            if (id_e > e_id && id_s < s_id) {
                not_int_range = true;
            }
        } else {
            if (id_s > e_id || id_e < s_id) {
                not_int_range = true;
            }
        }
    }
    if (not_int_range == false) {   // 不进行动态并行，线程不够用，一个SM才2048个线程
        if (edge_singl) {
            for (int i = id_s; i < max_ray_num; i++) {
                if (range_singl) {       // 超出range范围的不计算
                    if (isIdInRange<true>(s_id, e_id, i) == false) {
                        continue;
                    }
                } else {
                    if (isIdInRange<false>(s_id, e_id, i) == false) {
                        continue;
                    }
                }
                const double angle = ang_incre * static_cast<double>(i) - M_PI;
                const float rval = getRange(p1, vec_line, obs, angle);
                const int range_int  = floatToOrderedInt(rval);
                int offset = 0;
                if (range_singl) {
                    offset = getRangeOffset<true>(s_id, e_id, i, range_num);
                } else {
                    offset = getRangeOffset<false>(s_id, e_id, i, range_num);
                }
                int *pos = &range[offset];
                atomicMin(pos, range_int);         // 原子压入
            }
            for (int i = 0; i <= id_e; i++) {
                if (range_singl) {       // 超出range范围的不计算
                    if (isIdInRange<true>(s_id, e_id, i) == false) {
                        continue;
                    }
                } else {
                    if (isIdInRange<false>(s_id, e_id, i) == false) {
                        continue;
                    }
                }
                const double angle = ang_incre * static_cast<double>(i) - M_PI;
                const float rval = getRange(p1, vec_line, obs, angle);
                int range_int  = floatToOrderedInt(rval);
                int offset = 0;
                if (range_singl) {
                    offset = getRangeOffset<true>(s_id, e_id, i, range_num);
                } else {
                    offset = getRangeOffset<false>(s_id, e_id, i, range_num);
                }
                // if (offset >= range_num || offset < 0) {
                //     printf("Line 112, %d, %d, %d, %d\n", offset, range_singl, id_s, id_e);
                //     printf("%d, %d, %d, %d, %d, %f, %f\n", s_id, e_id, id_s, id_e, i, sang, eang);
                //     int test_res = 0;
                //     if (range_singl)        // 超出range范围的不计算
                //         test_res = (isIdInRange<true>(s_id, e_id, i) == false);
                //     else
                //         test_res = (isIdInRange<false>(s_id, e_id, i) == false);
                //     printf("Line 112, %d, %d, %d\n", id_s, id_e, test_res);
                // }
                int *pos = &range[offset];
                atomicMin(pos, range_int);         // 原子压入
            }
        } else {
            for (int i = id_s; i <= id_e; i++) {
                if (range_singl) {       // 超出range范围的不计算
                    if (isIdInRange<true>(s_id, e_id, i) == false) {
                        continue;
                    }
                } else {
                    if (isIdInRange<false>(s_id, e_id, i) == false) {
                        continue;
                    }
                }
                const double angle = ang_incre * static_cast<double>(i) - M_PI;
                const float rval = getRange(p1, vec_line, obs, angle);
                const int range_int  = floatToOrderedInt(rval);
                int offset = 0;
                if (range_singl) {
                    offset = getRangeOffset<true>(s_id, e_id, i, range_num);
                } else {
                    offset = getRangeOffset<false>(s_id, e_id, i, range_num);
                }
                int *pos = &range[offset];
                atomicMin(pos, range_int);         // 原子压入
            }
        }
    }
}

/// 共享内存需要用在flags / range上
__global__ void particleFilter(
    const Obsp* const ptcls,
    // const float* const raw_segs,
    const float* const ref, float* weights,
    const double ang_min, const double ang_incre, const int range_num, 
    const int full_rnum, const bool single_flag
) {
    extern __shared__ int range[];          //...一个数据类型分为了两个不同意义以及类型的块
    bool* flags = (bool*)(&range[range_num]);
    const int pid = blockIdx.x, sid = threadIdx.x;
    const Obsp* const obs_ptr = ptcls + pid;
    const Eigen::Vector2d this_obs(obs_ptr->x, obs_ptr->y);
    const double angle = obs_ptr->a;
    initialize(raw_segs, &this_obs, sid, flags);
    __syncthreads();
    const int s_id = static_cast<int>(ceil((ang_min + angle + M_PI) / ang_incre)) % full_rnum, 
        e_id = (s_id + range_num - 1) % full_rnum;
    const int th_num = blockDim.x;
    for (int i = 0; i < 4; i++) {       // 初始化深度（一个大值）
        const int cur_i = sid + i * th_num;
        if (cur_i >= range_num) {
            break;  // warp divergence 2
        }
        range[cur_i] = FLOAT_2048;
    }
    __syncthreads();
    if (flags[sid] == true) {           // warp divergence 1
        const float* const base = (raw_segs + 4 * sid);
        Eigen::Vector2d p1(*(base), *(base + 1)), p2(*(base + 2), *(base + 3));
        singleSegZbuffer(p1, p2, ptcls + pid, s_id, e_id, range_num, ang_incre, range);
    }
    __syncthreads();
    // 每个线程需要继续参与计算
    // 深度图计算完成之后，需要计算weight
    /// 每个block可以得到自己particle的深度图
    for (int i = 0; i < 4; i++) {       
        const int cur_i = sid + i * th_num;
        if (cur_i >= range_num) break;  // warp divergence 2
        if (single_flag) {
            weights[cur_i] = orderedIntToFloat(range[cur_i]);
        } else {
            float val = orderedIntToFloat(range[cur_i]);
            float abs_diff = abs(ref[cur_i] - val);
            float *pos = &weights[pid];
            atomicAdd(pos, abs_diff);
        }
    }
    __syncthreads();
    // 计算完每一个点的值
}

__global__ void initTest(
    const Obsp* const ptcls,
    bool* flags
) {
    const int pid = blockIdx.x, sid = threadIdx.x;
    const Obsp* const obs_ptr = ptcls + pid;
    const Eigen::Vector2d this_obs(obs_ptr->x, obs_ptr->y);
    const double angle = obs_ptr->a;
    initialize(raw_segs, &this_obs, sid, flags);
    __syncthreads();
}
