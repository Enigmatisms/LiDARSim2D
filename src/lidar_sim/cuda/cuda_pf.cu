#include "hip/hip_runtime.h"
#include <numeric>
#include <opencv2/imgproc.hpp>
// #include <opencv2/highgui.hpp>
#include <gnuplot-iostream.h>
#include "scanUtils.hpp"
#include "consts.h"
#include "cuda_pf.hpp"

const cv::Rect __walls(0, 0, 1200, 900);
const cv::Rect __floors(30, 30, 1140, 840);
constexpr int ALIGN_CHECK = 0x03;
constexpr double M_2PI = 2 * M_PI;

CudaPF::CudaPF(const cv::Mat& occ,  const Eigen::Vector3d& angles, int pnum, int resample_freq): 
    occupancy(occ), resample_freq(resample_freq), point_num(pnum * 72), 
    angle_min(angles(0)), angle_max(angles(1)), angle_incre(angles(2)), rng(0)
{
    ray_num = static_cast<int>(floor((angles(1) - angles(0)) / angle_incre));
    full_ray_num = std::round(2 * M_PI / angle_incre);
    seg_num = 0;
    
    #ifdef CUDA_CALC_TIME
    for (int i = 0; i < 5; i++) {
        time_sum[i] = 0.0;
        cnt_sum[i] = 0.0;
    }
    #endif // CALC_TIME
}

void CudaPF::particleInitialize(const cv::Mat& src, Eigen::Vector3d act_obs) {
    int pt_num = 0;
    particles.clear();
    // while (pt_num < point_num) { 
    //     const int x = rng.uniform(38, 1167);
    //     const int y = rng.uniform(38, 867);
    //     const double a = rng.uniform(0.0, M_2PI);
    //     if (src.at<uchar>(y, x) == 0x00) {
    //         particles.emplace_back(x, y, a);
    //         pt_num++;
    //     }
    // }
    while (pt_num < point_num) {
        const double x = rng.uniform(38, 1167);
        const double y = rng.uniform(38, 867);
        if (src.at<uchar>(y, x) > 0x00) continue;
        for (int i = 0; i < 180; i++) {
            const double dx = rng.gaussian(4);
            const double dy = rng.gaussian(4);
            particles.emplace_back(x + dx, y + dy, static_cast<double>(i) * M_PI / 36.0);
            pt_num++;
        }
    }
    // for (int i = 0; i < 10; i++) {
    //     const int x = act_obs.x() + rng.uniform(-4, 4);
    //     const int y = act_obs.y() + rng.uniform(-4, 4);
    //     const double a = act_obs.z() + rng.uniform(-0.1, 0.1);
    //     particles.emplace_back(x, y, a);
    // }
    CUDA_CHECK_RETURN(hipMemcpy(cu_pts, particles.data(), pt_num * sizeof(Obsp), hipMemcpyHostToDevice));
}

void CudaPF::particleUpdate(double mx, double my, double angle) {
    TicToc timer;
    timer.tic();
    for (Obsp& pt: particles) {
        double _mx = mx, _my = my, _a = angle;
        noisedMotion(_mx, _my, _a);
        pt.x += _mx;
        pt.y += _my;
        pt.a += _a;
        if (pt.a < 0)
            pt.a += M_2PI;
        else if (pt.a > M_2PI)
            pt.a -= M_2PI;
    }
    time_sum[2] += timer.toc();
    cnt_sum[2] += 1.0;
    timer.tic();
    CUDA_CHECK_RETURN(hipMemcpy(cu_pts, particles.data(), particles.size() * sizeof(Obsp), hipMemcpyHostToDevice));
    time_sum[3] += timer.toc();
    cnt_sum[3] += 1.0;
}

__host__ void CudaPF::intialize(const std::vector<std::vector<cv::Point>>& obstacles) {
    CUDA_CHECK_RETURN(hipMalloc((void **) &weights, point_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &ref_range, ray_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_pts, point_num * sizeof(Obsp)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &obs, sizeof(Obsp)));

    // 首先计算segment
    std::vector<std::vector<Eigen::Vector2d>> obstcs;
    for (const std::vector<cv::Point>& obstacle: obstacles) {            // 构建objects
        obstcs.emplace_back();
        for (const cv::Point& pt: obstacle)
            obstcs.back().emplace_back(pt.x, pt.y);
    }
    // ================ add walls ================
    obstcs.emplace_back();
    for (const cv::Point& pt: north_wall)
        obstcs.back().emplace_back(pt.x, pt.y); 
    obstcs.emplace_back();
    for (const cv::Point& pt: east_wall)
        obstcs.back().emplace_back(pt.x, pt.y); 
    obstcs.emplace_back();
    for (const cv::Point& pt: south_wall)
        obstcs.back().emplace_back(pt.x, pt.y); 
    obstcs.emplace_back();
    for (const cv::Point& pt: west_wall)
        obstcs.back().emplace_back(pt.x, pt.y);
    seg_num = 0;
    for (const std::vector<Eigen::Vector2d>& obst: obstcs) {
        for (size_t i = 1; i < obst.size(); i++) {
            const Eigen::Vector2d& p = obst[i - 1];
            const Eigen::Vector2d& q = obst[i];
            host_seg.push_back(p.x());
            host_seg.push_back(p.y());
            host_seg.push_back(q.x());
            host_seg.push_back(q.y());
            seg_num ++;
        }
        const Eigen::Vector2d& p = obst.back();
        const Eigen::Vector2d& q = obst.front();
        host_seg.push_back(p.x());
        host_seg.push_back(p.y());
        host_seg.push_back(q.x());
        host_seg.push_back(q.y());
        seg_num ++;
    }
    copyRawSegs(host_seg.data(), sizeof(float) * host_seg.size());
    shared_to_allocate = sizeof(float) * ray_num + sizeof(bool) * seg_num;
    const int check_result = (shared_to_allocate & ALIGN_CHECK);
    if (check_result > 0)
        shared_to_allocate = shared_to_allocate + 4 - check_result;
    printf("Host segnum: %lu, first two: %f, %f\n", host_seg.size(), host_seg[0], host_seg[1]);
    // 分配4的整数个字节 才能保证初始float数组的完整性
}

Gnuplot gp1;
void CudaPF::filtering(const std::vector<std::vector<cv::Point>>& obstacles, Eigen::Vector3d act_obs, cv::Mat& src) {
    static int sampler_cnt = 0;
    TicToc timer;
    cv::rectangle(src, __walls, cv::Scalar(10, 10, 10), -1);
    cv::rectangle(src, __floors, cv::Scalar(40, 40, 40), -1);
    cv::drawContours(src, obstacles, -1, cv::Scalar(10, 10, 10), -1);
    // act_obs 的 z是角度，但是必须要0-2pi
    std::vector<float> weight_vec(point_num, 1.0 / static_cast<float>(point_num));
    if (sampler_cnt == 0) {
        Obsp host_obs(act_obs(0), act_obs(1), (act_obs.z() < 0) ? act_obs.z() + M_2PI : act_obs.z());
        timer.tic();
        CUDA_CHECK_RETURN(hipMemcpy(obs, &host_obs, sizeof(Obsp), hipMemcpyHostToDevice));
        particleFilter <<< 1, seg_num, shared_to_allocate >>> (
                            obs, NULL, ref_range, angle_min, angle_incre, ray_num, full_ray_num, true);
        particleFilter <<< point_num, seg_num, shared_to_allocate >>> (
                            cu_pts, ref_range, weights, angle_min, angle_incre, ray_num, full_ray_num, false);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
        time_sum[0] += timer.toc();
        cnt_sum[0] += 1.0;
        timer.tic();
        CUDA_CHECK_RETURN(hipMemcpy(weight_vec.data(), weights, sizeof(float) * point_num, hipMemcpyDeviceToHost));

        #pragma omp parallel for num_threads(8)
        for (int i = 0; i < point_num; i++) {
            const Obsp& pt = particles[i];
            weight_vec[i] /= static_cast<float>(ray_num);
            const int ptx = pt.x, pty = pt.y;
            if (ptx < 0 || pty < 0 || ptx >= 1200 || pty >= 900) {
                weight_vec[i] *= 1.5;
            } else {
                if (occupancy.at<uchar>(pty, ptx) > 0x00) {
                    weight_vec[i] *= 1.5;
                }
            }
            weight_vec[i] = 1.0 / (weight_vec[i] + 1.0);
        }

        float weight_sum = std::accumulate(weight_vec.begin(), weight_vec.end(), 0.0);
        for (float& val: weight_vec)                                  // 归一化形成概率
            val /= weight_sum;
        gp1 << "plot" << gp1.file1d(weight_vec) << "with line title 'range1'\n" << std::endl;

        importanceResampler(weight_vec);                               // 重采样
    }
    sampler_cnt = (sampler_cnt + 1) % resample_freq;
    time_sum[1] += timer.toc();
    cnt_sum[1] += 1.0;
    visualizeParticles(weight_vec, src);
    cv::circle(src, cv::Point(act_obs.x(), act_obs.y()), 5, cv::Scalar(0, 255, 255), -1);
}

/// @ref implementation from Thrun: Probabilistic Robotics
void CudaPF::importanceResampler(const std::vector<float>& weights) {
    std::vector<Obsp> tmp;
    std::vector<int> tmp_ids;
    double dpoint_num = static_cast<double>(point_num);
    double r = rng.uniform(0.0, 1.0 / dpoint_num);
    double c = weights.front();
    int i = 0;
    for (int m = 1; m <= point_num; m++) {
        double u = r + static_cast<double>(m - 1) / dpoint_num;
        while (u > c) {
            i++;
            c += weights[i];
        }
        Obsp new_p;
        new_p.x = particles[i].x + rng.gaussian(0.5);
        new_p.y = particles[i].y + rng.gaussian(0.5);
        new_p.a = particles[i].a + rng.gaussian(0.04);
        tmp.push_back(new_p);
    }
    particles.assign(tmp.begin(), tmp.end());
}

void CudaPF::scanPerturb(std::vector<float>& range) {
    for (float& val: range)
        val += rng.gaussian(7);
}

void CudaPF::visualizeParticles(const std::vector<float>& weights, cv::Mat& dst) const {
    Eigen::Vector2d center;
    double weight_sum = 0.0;
    center.setZero();
    for (size_t i = 0; i < particles.size(); i++) {
        const Obsp& pt = particles[i];
        center += weights[i] * Eigen::Vector2d(pt.x, pt.y);
        weight_sum += weights[i];
        double val = weights[i];
        uchar color_val = 254.0 * val, inv_color_val = 255.0 - color_val;
        cv::Scalar color(color_val, 0, inv_color_val);
        cv::circle(dst, cv::Point(pt.x, pt.y), 3, color, -1);
    }
    cv::circle(dst, cv::Point(center.x(), center.y()), 4, cv::Scalar(255, 0, 0), -1);
}

void CudaPF::singleDebugDemo(const std::vector<std::vector<cv::Point>>& obstacles, Eigen::Vector3d act_obs, cv::Mat& src) {
    cv::rectangle(src, cv::Rect(0, 0, 1200, 900), cv::Scalar(10, 10, 10), -1);
    cv::rectangle(src, cv::Rect(30, 30, 1140, 840), cv::Scalar(40, 40, 40), -1);
    cv::drawContours(src, obstacles, -1, cv::Scalar(10, 10, 10), -1);
    act_obs(2) = (act_obs.z() < 0) ? act_obs.z() + M_2PI : act_obs.z();
    int start_id = static_cast<int>(ceil((angle_min + act_obs(2) + M_PI) / angle_incre)) % full_ray_num, 
        end_id = (start_id + ray_num - 1) % full_ray_num;
    Obsp host_obs(act_obs(0), act_obs(1), act_obs(2));
    TicToc timer;
    timer.tic();
    CUDA_CHECK_RETURN(hipMemcpy(obs, &host_obs, sizeof(Obsp), hipMemcpyHostToDevice));
    particleFilter <<< 1, seg_num, shared_to_allocate >>> (
                        obs, NULL, ref_range, angle_min, angle_incre, ray_num, full_ray_num, true);
    std::vector<float> range(ray_num, 0.0);
    std::cout << "Time consumption:" << timer.toc() << std::endl;
    CUDA_CHECK_RETURN(hipMemcpy(range.data(), ref_range, sizeof(float) * ray_num, hipMemcpyDeviceToHost));
    // importanceResampler(weight_vec);                               // 重采样
    const cv::Point cv_obs(act_obs.x(), act_obs.y());
    for (int i = 0; i < ray_num; i++) {
        double angle = static_cast<double>(i + start_id) * angle_incre - M_PI;
        double rval = range[i];
        cv::Point2d trans(rval * cos(angle), rval * sin(angle));
        cv::Point lpt = cv_obs + cv::Point(trans.x, trans.y);
        cv::line(src, cv_obs, lpt, cv::Scalar(0, 0, 255), 1);
    }
}

void CudaPF::edgeDispDemo(const std::vector<std::vector<cv::Point>>& obstacles, Eigen::Vector3d act_obs, cv::Mat& src) {
        cv::rectangle(src, cv::Rect(0, 0, 1200, 900), cv::Scalar(10, 10, 10), -1);
    cv::rectangle(src, cv::Rect(30, 30, 1140, 840), cv::Scalar(40, 40, 40), -1);
    cv::drawContours(src, obstacles, -1, cv::Scalar(10, 10, 10), -1);
    int start_id = static_cast<int>(ceil((angle_min + act_obs(2) + M_PI) / angle_incre)) % full_ray_num, 
        end_id = (start_id + ray_num - 1) % full_ray_num;
    Obsp host_obs(act_obs(0), act_obs(1), (act_obs.z() < 0) ? act_obs.z() + M_2PI : act_obs.z());
    bool* host_flag = new bool[seg_num];
    bool* flags;
    CUDA_CHECK_RETURN(hipMalloc((void **) &flags, seg_num * sizeof(bool)));
    CUDA_CHECK_RETURN(hipMemcpy(obs, &host_obs, sizeof(Obsp), hipMemcpyHostToDevice));
    initTest <<< 1, seg_num >>> (obs, flags);
    CUDA_CHECK_RETURN(hipMemcpy(host_flag, flags, sizeof(bool) * seg_num, hipMemcpyDeviceToHost));
    for (size_t i = 0; i < size_t(seg_num); i++) {
        if (host_flag[i] == true) {
            const size_t base = 4 * i;
            cv::Point p1(host_seg[base], host_seg[base + 1]);
            cv::Point p2(host_seg[base + 2], host_seg[base + 3]);
            cv::line(src, p1, p2, cv::Scalar(0, 255, 0), 2);
        }
    }
    cv::circle(src, cv::Point(act_obs.x(), act_obs.y()), 3, cv::Scalar(0, 0, 255), -1);
    CUDA_CHECK_RETURN(hipFree(flags));
    delete [] host_flag;
}

void CudaPF::pfTestDeom(const std::vector<std::vector<cv::Point>>& obstacles, Eigen::Vector3d act_obs, cv::Mat& src) {
    ;
}


    // #pragma omp parallel for num_threads(8)
    // for (int i = 0; i < point_num; i++) {
    //     const Obsp& pt = particles[i];
    //     weight_vec[i] /= static_cast<float>(ray_num);
    //     const int ptx = pt.x, pty = pt.y;
    //     if (ptx < 0 || pty < 0 || ptx >= 1200 || pty >= 900) {
    //         weight_vec[i] = 0.0;
    //     } else {
    //         if (occupancy.at<uchar>(pty, ptx) > 0x00) {
    //             weight_vec[i] = 0.0;
    //         } else {
    //             weight_vec[i] = 1.0 / (weight_vec[i] + 1.0);
    //         }
    //     }
    //     weight_vec[i] = 1.0 / (weight_vec[i] + 1.0);
    // }