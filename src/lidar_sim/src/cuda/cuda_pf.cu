#include "hip/hip_runtime.h"
#include <numeric>
#include <opencv2/imgproc.hpp>
#include <hip/hip_runtime_api.h>
#include "utils/scanUtils.hpp"
#include "utils/consts.h"
#include "cuda/cuda_pf.hpp"

const cv::Rect __walls(0, 0, 1200, 900);
const cv::Rect __floors(30, 30, 1140, 840);
constexpr int ALIGN_CHECK = 0x03;
constexpr double M_2PI = 2 * M_PI;

CudaPF::CudaPF(const cv::Mat& occ,  const Eigen::Vector3d& angles, int pnum, std::string path): 
    occupancy(occ), point_num(pnum * 10), cascade_num(point_num >> 5),
    angle_min(angles(0)), angle_max(angles(1)), angle_incre(angles(2)), rng(0)
{
    ray_num = static_cast<int>(floor((angles(1) - angles(0)) / angle_incre));
    full_ray_num = std::round(2 * M_PI / angle_incre);
    seg_num = 0;
    weight_vec.resize(point_num, 1.0 / static_cast<float>(point_num));
    #ifdef CUDA_CALC_TIME
    for (int i = 0; i < 5; i++) {
        time_sum[i] = 0.0;
        cnt_sum[i] = 0.0;
    }
    #endif // CALC_TIME
    #ifdef SAVE_RANGE_FILE
        file.open(path, std::ios::out);
    #endif
}

void CudaPF::particleInitialize(const cv::Mat& src, Eigen::Vector3d act_obs) {
    int pt_num = 0;
    while (pt_num < point_num) {
        const double x = rng.uniform(38, 1167);
        const double y = rng.uniform(38, 867);
        if (src.at<uchar>(y, x) > 0x00) continue;
        for (int i = 0; i < 10; i++) {
            const double dx = rng.gaussian(2);
            const double dy = rng.gaussian(2);
            particles[pt_num] = Obsp(x + dx, y + dy, static_cast<double>(i) * M_PI / 5.0);
            pt_num++;
        }
    }
    CUDA_CHECK_RETURN(hipMemcpy(cu_pts, particles, point_num * sizeof(Obsp), hipMemcpyHostToDevice));
}

void CudaPF::particleUpdate(const Eigen::Vector3d& act_obs, double mx, double my, double angle) {
    TicToc timer;
    timer.tic();
    #pragma omp parallel for num_threads(8)
    for (int i = 0; i < point_num; i++) {
        Obsp& pt = particles[i];
        const double cosa = cos(pt.a), sina = sin(pt.a);
        pt.x += cosa * mx + sina * my;
        pt.y += sina * mx - cosa * my;
        pt.a += angle;
    }
    for (int i = 0; i < point_num; i++) {
        Obsp& pt = particles[i];
        double _mx = 0, _my = 0, _a = 0;
        noisedMotion(_mx, _my, _a);
        pt.x += _mx;
        pt.y += _my;
        pt.a += _a;
        if (pt.a < 0)
            pt.a += M_2PI;
        else if (pt.a > M_2PI)
            pt.a -= M_2PI;
    }
    // act_obs 的 z是角度，但是必须要0-2pi
    obs->x = act_obs.x();
    obs->y = act_obs.y();
    obs->a = (act_obs.z() < 0) ? act_obs.z() + M_2PI : act_obs.z();
    time_sum[2] += timer.toc();
    cnt_sum[2] += 1.0;
    timer.tic();
    CUDA_CHECK_RETURN(hipMemcpy(cu_pts, particles, (point_num + 1) * sizeof(Obsp), hipMemcpyHostToDevice));
    time_sum[3] += timer.toc();
    cnt_sum[3] += 1.0;
}

__host__ void CudaPF::intialize(const std::vector<std::vector<cv::Point>>& obstacles) {
    CUDA_CHECK_RETURN(hipMalloc((void **) &weights, point_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &ref_range, ray_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &cu_pts, (point_num + 1) * sizeof(Obsp)));
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &particles, (point_num + 1) * sizeof(Obsp)));
    obs = &particles[point_num];

    // 首先计算segment
    std::vector<std::vector<Eigen::Vector2d>> obstcs;
    for (const std::vector<cv::Point>& obstacle: obstacles) {            // 构建objects
        obstcs.emplace_back();
        for (const cv::Point& pt: obstacle)
            obstcs.back().emplace_back(pt.x, pt.y);
    }
    // ================ add walls ================
    obstcs.emplace_back();
    for (const cv::Point& pt: north_wall)
        obstcs.back().emplace_back(pt.x, pt.y); 
    obstcs.emplace_back();
    for (const cv::Point& pt: east_wall)
        obstcs.back().emplace_back(pt.x, pt.y); 
    obstcs.emplace_back();
    for (const cv::Point& pt: south_wall)
        obstcs.back().emplace_back(pt.x, pt.y); 
    obstcs.emplace_back();
    for (const cv::Point& pt: west_wall)
        obstcs.back().emplace_back(pt.x, pt.y);
    seg_num = 0;
    for (const std::vector<Eigen::Vector2d>& obst: obstcs) {
        for (size_t i = 1; i < obst.size(); i++) {
            const Eigen::Vector2d& p = obst[i - 1];
            const Eigen::Vector2d& q = obst[i];
            host_seg.push_back(p.x());
            host_seg.push_back(p.y());
            host_seg.push_back(q.x());
            host_seg.push_back(q.y());
            seg_num ++;
        }
        const Eigen::Vector2d& p = obst.back();
        const Eigen::Vector2d& q = obst.front();
        host_seg.push_back(p.x());
        host_seg.push_back(p.y());
        host_seg.push_back(q.x());
        host_seg.push_back(q.y());
        seg_num ++;
    }
    copyRawSegs(host_seg.data(), sizeof(float) * host_seg.size());
    shared_to_allocate = sizeof(float) * ray_num + sizeof(bool) * seg_num;
    const int check_result = (shared_to_allocate & ALIGN_CHECK);
    if (check_result > 0)
        shared_to_allocate = shared_to_allocate + 4 - check_result;
    printf("Host segnum: %lu, first two: %f, %f\n", host_seg.size(), host_seg[0], host_seg[1]);
    // 分配4的整数个字节 才能保证初始float数组的完整性
}

void CudaPF::filtering(const std::vector<std::vector<cv::Point>>& obstacles, Eigen::Vector3d act_obs, cv::Mat& src) {
    TicToc timer;
    timer.tic();
    // hipProfilerStart();
    particleFilter <<< 1, seg_num, shared_to_allocate >>> (
                        &cu_pts[point_num], NULL, ref_range, angle_min, angle_incre, ray_num, full_ray_num, 0, true);
    hipStream_t streams[8];
    for (int i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    for (int i = 0; i < cascade_num; i++) {
        particleFilter <<< 32, seg_num, shared_to_allocate, streams[i % 8]>>> (
                    cu_pts, ref_range, weights, angle_min, angle_incre, ray_num, full_ray_num, i << 5, false);
    }
    #ifdef SAVE_RANGE_FILE
    std::vector<float> range(ray_num, 0.0);
    CUDA_CHECK_RETURN(hipMemcpy(range.data(), ref_range, sizeof(float) * ray_num, hipMemcpyDeviceToHost));
    int start_id = static_cast<int>(ceil((angle_min + act_obs(2) + M_PI) / angle_incre)) % full_ray_num;
    file << act_obs(0) << "," << act_obs(1) << "," << start_id << ",";
    for (int i = 0; i < ray_num; i++) {
        file << range[i] << ",";
    }
    file << std::endl;
    #endif
    cv::rectangle(src, __walls, cv::Scalar(10, 10, 10), -1);
    cv::rectangle(src, __floors, cv::Scalar(40, 40, 40), -1);
    cv::drawContours(src, obstacles, -1, cv::Scalar(10, 10, 10), -1);
    cv::circle(src, cv::Point(act_obs.x(), act_obs.y()), 5, cv::Scalar(0, 255, 255), -1);
    visualizeParticles(weight_vec, src);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // hipProfilerStop();
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    time_sum[0] += timer.toc();
    cnt_sum[0] += 1.0;
    CUDA_CHECK_RETURN(hipMemcpy(weight_vec.data(), weights, sizeof(float) * point_num, hipMemcpyDeviceToHost));

    #pragma omp parallel for num_threads(8)
    for (int i = 0; i < point_num; i++) {
        const Obsp& pt = particles[i];
        weight_vec[i] /= static_cast<float>(ray_num);
        const int ptx = pt.x, pty = pt.y;
        if (ptx < 0 || pty < 0 || ptx >= 1200 || pty >= 900) {
            weight_vec[i] *= 1.5;
        } else {
            if (occupancy.at<uchar>(pty, ptx) > 0x00) {
                weight_vec[i] *= 1.5;
            }
        }
        weight_vec[i] = 1.0 / (weight_vec[i] + 1.0);
    }
    timer.tic();
    float weight_sum = std::accumulate(weight_vec.begin(), weight_vec.end(), 0.0);
    for (float& val: weight_vec)                                  // 归一化形成概率
        val /= weight_sum;
    importanceResampler(weight_vec);                               // 重采样
    time_sum[1] += timer.toc();
    cnt_sum[1] += 1.0;
}

/// @ref implementation from Thrun: Probabilistic Robotics
void CudaPF::importanceResampler(const std::vector<float>& weights) {
    std::vector<Obsp> tmp;
    std::vector<int> tmp_ids;
    double dpoint_num = static_cast<double>(point_num);
    double r = rng.uniform(0.0, 1.0 / dpoint_num);
    double c = weights.front();
    int i = 0;
    for (int m = 1; m <= point_num; m++) {
        double u = r + static_cast<double>(m - 1) / dpoint_num;
        while (u > c) {
            i++;
            c += weights[i];
        }
        tmp.push_back(particles[i]);
    }
    #pragma omp parallel for num_threads(2)
    for (size_t i = 0; i < tmp.size(); i++)
        particles[i] = tmp[i];
}

void CudaPF::scanPerturb(std::vector<float>& range) {
    for (float& val: range)
        val += rng.gaussian(7);
}

void CudaPF::visualizeParticles(const std::vector<float>& weights, cv::Mat& dst) const {
    Eigen::Vector2d center;
    center.setZero();
    for (int i = 0; i < point_num; i++) {
        const Obsp& pt = particles[i];
        center += weights[i] * Eigen::Vector2d(pt.x, pt.y);
        const cv::Scalar color(0, 0, 255);
        cv::circle(dst, cv::Point(pt.x, pt.y), 3, color, -1);
    }
    cv::circle(dst, cv::Point(center.x(), center.y()), 4, cv::Scalar(255, 0, 0), -1);
}

void CudaPF::singleDebugDemo(const std::vector<std::vector<cv::Point>>& obstacles, Eigen::Vector3d act_obs, cv::Mat& src) {
    cv::rectangle(src, cv::Rect(0, 0, 1200, 900), cv::Scalar(10, 10, 10), -1);
    cv::rectangle(src, cv::Rect(30, 30, 1140, 840), cv::Scalar(40, 40, 40), -1);
    cv::drawContours(src, obstacles, -1, cv::Scalar(10, 10, 10), -1);
    act_obs(2) = (act_obs.z() < 0) ? act_obs.z() + M_2PI : act_obs.z();
    int start_id = static_cast<int>(ceil((angle_min + act_obs(2) + M_PI) / angle_incre)) % full_ray_num;
    Obsp host_obs(act_obs(0), act_obs(1), act_obs(2));
    TicToc timer;
    timer.tic();
    CUDA_CHECK_RETURN(hipMemcpy(obs, &host_obs, sizeof(Obsp), hipMemcpyHostToDevice));
    particleFilter <<< 1, seg_num, shared_to_allocate >>> (
                        obs, NULL, ref_range, angle_min, angle_incre, ray_num, full_ray_num, 0, true);
    std::vector<float> range(ray_num, 0.0);
    std::cout << "Time consumption:" << timer.toc() << std::endl;
    CUDA_CHECK_RETURN(hipMemcpy(range.data(), ref_range, sizeof(float) * ray_num, hipMemcpyDeviceToHost));
    const cv::Point cv_obs(act_obs.x(), act_obs.y());
    for (int i = 0; i < ray_num; i++) {
        double angle = static_cast<double>(i + start_id) * angle_incre - M_PI;
        double rval = range[i];
        cv::Point2d trans(rval * cos(angle), rval * sin(angle));
        cv::Point lpt = cv_obs + cv::Point(trans.x, trans.y);
        cv::line(src, cv_obs, lpt, cv::Scalar(0, 0, 255), 1);
    }
}
