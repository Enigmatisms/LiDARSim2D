#include "hip/hip_runtime.h"
#include <numeric>
#include "cuda/host_func.hpp"
#include "utils/scanUtils.hpp"

short *sid_ptr = nullptr, *eid_ptr = nullptr;
float *all_segments = nullptr, *angles_ptr = nullptr, *dists_ptr = nullptr, *final_dense_ranges, *final_sparse_ranges, *oct_ranges;
bool *flag_ptr = nullptr;
size_t total_seg_num = 0;

__host__ void intializeFixed(int num_ray) {
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &all_segments, 8192 * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &final_dense_ranges, num_ray * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &final_sparse_ranges, num_ray / 3 * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &oct_ranges, (num_ray << 3) * sizeof(float)));
}

__host__ void deallocateFixed() {
    CUDA_CHECK_RETURN(hipHostFree(all_segments));
    CUDA_CHECK_RETURN(hipFree(final_dense_ranges));
    CUDA_CHECK_RETURN(hipFree(final_sparse_ranges));
    CUDA_CHECK_RETURN(hipFree(oct_ranges));
}

__host__ void deallocateDevice() {
    CUDA_CHECK_RETURN(hipFree(sid_ptr));
    CUDA_CHECK_RETURN(hipFree(eid_ptr));
    CUDA_CHECK_RETURN(hipFree(angles_ptr));
    CUDA_CHECK_RETURN(hipFree(dists_ptr));
    CUDA_CHECK_RETURN(hipFree(flag_ptr));
}

__host__ void unwrapMeshes(const Meshes& meshes, bool initialized) {
    size_t mesh_point_cnt = 0;
    total_seg_num = 0;
    for (const Mesh& m: meshes) {
        size_t max_size = m.size() - 1;
        all_segments[mesh_point_cnt++] = m.front().x();
        all_segments[mesh_point_cnt++] = m.front().y();
        for (size_t i = 1; i < max_size; i++) {
            const Eigen::Vector2d& p = m[i];
            float x = p.x(), y = p.y();
            all_segments[mesh_point_cnt++] = x;
            all_segments[mesh_point_cnt++] = y;
            all_segments[mesh_point_cnt++] = x;
            all_segments[mesh_point_cnt++] = y;
        }
        all_segments[mesh_point_cnt++] = m.back().x();
        all_segments[mesh_point_cnt++] = m.back().y();
        total_seg_num += (m.size() - 1);
    }
    updateSegments(all_segments, mesh_point_cnt << 2);
    if (initialized == true)
        deallocateDevice();

    CUDA_CHECK_RETURN(hipMalloc((void **) &sid_ptr, total_seg_num * sizeof(short)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &eid_ptr, total_seg_num * sizeof(short)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &angles_ptr, total_seg_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &dists_ptr, total_seg_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &flag_ptr, total_seg_num * sizeof(bool)));
}

// 外部判定，如果激光线数不是120的整数倍，则报错（所求的深度图将是线数 * 3，由于需要模拟深度不连续）
__host__ double rayTraceRenderCpp(const Eigen::Vector3d& lidar_param, const Eigen::Vector2d& pose, float angle, int ray_num, std::vector<float>& range) {
    // 对于静态地图而言，由于场景无需频繁update，unwrapMeshes函数调用频率低，则可以省略内存allocation操作
    const int lidar_ray_blocks = ray_num / DEPTH_DIV_NUM;
    const short num_blocks = static_cast<short>(ceilf(total_seg_num / 256.f));          // 面片数 / 128
    TicToc timer;
    timer.tic();
    preProcess<<<num_blocks, 256>>>(sid_ptr, eid_ptr, angles_ptr, dists_ptr, flag_ptr, ray_num, 
                total_seg_num, lidar_param.x(), lidar_param.z(), pose.x(), pose.y(), angle);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    double result = timer.toc();
    hipStream_t streams[8];
    for (short i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    const short segment_per_block = static_cast<short>(ceil(0.125f * total_seg_num)),
                last_block_seg_num = short(total_seg_num) - 7 * segment_per_block;
    const size_t size_t_sblock = static_cast<size_t>(segment_per_block);
    const size_t shared_mem_size = (size_t_sblock * 13) + (DEPTH_DIV_NUM << 2) + 4 - size_t_sblock % 4;     // 13 = 8 + 4 + 1 = (4B angles 4B dists, 2B * 2 ids, 1B flags)
    for (int i = 0; i < 8; i++) {
        // 最后由于bool是单字节的类型，需要padding到4的整数倍字节数
        // local segments大小应该是 4B * (angles + dists) / 8 + DEPTH_DIV_NUM * 4B (深度图分区) + 2B * (sids + eids) / 8 + (1B * len(flags) / 8) + padding
        rayTraceKernel<<<lidar_ray_blocks, DEPTH_DIV_NUM, shared_mem_size, streams[i]>>>(
            sid_ptr, eid_ptr, angles_ptr, dists_ptr, flag_ptr, i, segment_per_block, 
            ((i < 7) ? segment_per_block : last_block_seg_num), &oct_ranges[i * ray_num], lidar_param.x(), lidar_param.z(), angle
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    getMininumRangeKernel<<<lidar_ray_blocks, DEPTH_DIV_NUM>>>(oct_ranges, final_dense_ranges, ray_num);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    sparsifyScan<<<lidar_ray_blocks, DEPTH_DIV_NUM / 3>>>(final_dense_ranges, final_sparse_ranges);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // 注意range的大小应该提前确定
    CUDA_CHECK_RETURN(hipMemcpy(range.data(), final_sparse_ranges, sizeof(float) * ray_num / 3, hipMemcpyDeviceToHost));
    return result;
}
